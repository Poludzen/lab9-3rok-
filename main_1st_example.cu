
#include <hip/hip_runtime.h>
#include "cstdio"
#include "ctime"

__global__ void GPU(const double* d, double* out){
    int ind = blockDim.x * blockIdx.x + threadIdx.x;
    for(int i = 1; i < d[ind]; i++){
        out[ind] += i;
    }
}

// CPU example with showing time of execution
void testCPU(int count, const double* arr){
    clock_t start = clock();
    auto* out = new double[count];
    for(int j = 0; j < count; j++){
        for(int i = 1; i < arr[j]; i++){
            out[j] += i;
        }
    }
    clock_t end = clock();
    double seconds = (double)(end - start) / CLOCKS_PER_SEC;
    printf("The time: %f seconds\n", seconds);
}

// GPU example with showing time of execution
void testGPU(int count, const double* arr){
    clock_t start = clock();
    double* aa, *out_;
    hipMalloc(&aa, sizeof(double) * count);
    hipMalloc(&out_, sizeof(double) * count);

    hipMemcpy(aa, arr, sizeof(double) * count, hipMemcpyHostToDevice);
    GPU<<<count / 100,100>>>(aa, out_);
    auto* out_c = (double*)malloc(sizeof(double) * count);
    hipMemcpy(out_c, out_, sizeof(double ) * count, hipMemcpyDeviceToHost);
    clock_t end = clock();
    double seconds = (double)(end - start) / CLOCKS_PER_SEC;
    printf("The time: %f seconds\n", seconds);

    hipFree(aa);
    hipFree(out_);
}
int main()
{
    // starting experiment 
    int count = 1000000;
    auto* arr = new double[count];
    for(int i = 0; i < count; i++){
        arr[i] = 100000;
    }
    // testing
    //testCPU(count, arr);
    testGPU(count, arr);



    return 0;
}

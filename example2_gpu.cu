#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include "cstdio"
#include "ctime"

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int counts[] = { 500000,1000000,2500000,5000000,10000000,50000000,100000000 };
  for(int j = 0;j<7;j++){
    int N = counts[j];
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
      x[i] = 1.0f;
      y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
    clock_t start = clock();
    // Perform SAXPY on N elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
    clock_t end = clock();
    double seconds = (double)(end - start) / CLOCKS_PER_SEC;
    printf("The time: %f seconds\n", seconds);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
      maxError = max(maxError, abs(y[i]-4.0f));
    
    //printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
  }
}
